#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>

// Cuda libraries
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// Cuda random number generators
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <time.h>
#include <stdlib.h>


__device__ float generate(hiprandState* globalState, int ind)
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

__global__ void addToCount(int N, int *y, hiprandState* globalState)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    while (id < blockDim.x * gridDim)
    {
        int number = generate(globalState, id) * 1000000;
        printf("%i\n", number);

        atomicAdd(&(y[0]), number);
        id += blockDim.x * gridDim.x;
    }
}

int main(void)
{
  int thread_count = 5;
  int *y, *d_y;
  y = (int*)malloc(thread_count*sizeof(int));

  hipMalloc(&d_y, thread_count * sizeof(int));
  hipMemcpy(d_y, y, thread_count * sizeof(int), hipMemcpyHostToDevice);

  hiprandState* devStates;
  hipMalloc (&devStates, thread_count * sizeof(hiprandState));
  srand(time(0));
  int seed = rand();

  setup_kernel<<<2, thread_count>>>(devStates,seed);
  addToCount<<<2, thread_count>>>(thread_count, d_y, devStates);

  hipMemcpy(y, d_y, thread_count * sizeof(int), hipMemcpyDeviceToHost);
  printf("%i\n", *y);
}


// sets up the devices and runs
extern "C" void setup_and_run(int myrank, grid_size, int thread_count)
{
    // Set device to the rank
    int cudaDeviceCount;
    hipError_t cE; 
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount );
        exit(-1);
    }
    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess )
    {
        printf(" Unable to have myrank %d set to cuda device %d, error is %d \n", myrank, (myrank % cudaDeviceCount), cE);
        exit(-1); 
    }
}

// copy memory from device to host
extern "C" void device_to_host(double * device_pointer, double * host_pointer, unsigned int size)
{
    hipMemcpy(host_pointer, device_pointer, size, hipMemcpyDeviceToHost);
}

// copy memory from host to device
extern "C" void host_to_device(double * host_pointer, double * device_pointer, unsigned int size)
{
    hipMemcpy(device_pointer, host_pointer, size, hipMemcpyHostToDevice);
}

extern "C" void colony_kernelLaunch( unsigned char** d_data, unsigned char** d_resultData, 
        int block_count, int thread_count, 
        unsigned int worldWidth, unsigned int worldHeight, 
        int myrank){

    // Call the kernel
    // HL_kernel<<<block_count,thread_count>>>(*d_data, *d_resultData, worldWidth, worldHeight);
    hipDeviceSynchronize();
}


extern "C" void freeCuda(double* ptr){
    hipFree(ptr);
}