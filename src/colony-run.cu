#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



// copy memory from device to host
extern "C" void host_to_device(float * device_pointer, float * host_pointer unsigned int size)
{
    hipMemcpy(host_pointer, device_pointer, size, hipMemcpyDeviceToHost);
}

// copy memory from host to device
extern "C" void host_to_device(float * host_pointer, float * device_pointer unsigned int size)
{
    hipMemcpy(device_pointer, host_pointer, size, hipMemcpyHostToDevice);
}

extern "C" void HL_kernelLaunch( unsigned char** d_data, unsigned char** d_resultData, 
        int block_count, int thread_count, 
        unsigned int worldWidth, unsigned int worldHeight, 
        int myrank){

    // Call the kernel
    HL_kernel<<<block_count,thread_count>>>(*d_data, *d_resultData, worldWidth, worldHeight);
    hipDeviceSynchronize();
}


extern "C" void freeCuda(float* ptr){
    hipFree(ptr);
}